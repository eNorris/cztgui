#include "hip/hip_runtime.h"

#include <assert.h>

#include <hip/hip_runtime.h>

#include "cuda_common/inc/helper_functions.h"
#include "cuda_common/inc/hip/hip_runtime_api.h"
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#include <iostream>
#include <ctime>

#include "./cudakernel.h"

using namespace std;

int main()
{

    // Rand seed for consistency
    srand(0);
    std::clock_t start;
    int simcycles = 10000;

    int devID;
    hipDeviceProp_t props;

    int devCount;
    hipGetDeviceCount(&devCount);
    cout << "Cuda devices: " << devCount << endl;

    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    cout << "Device " << devID << ": " << props.name << " with compute " 
         << props.major << "." << props.minor << " capability" << endl;

    start = std::clock();
    float *cpu_data;
    int nx = 1024, ny = 1024;

    // Work with streams
    int *hp = new int, *lp = new int;
    hipDeviceGetStreamPriorityRange(lp, hp);
    cout << "HP = " << *hp << "   LP = " << *lp << endl;

    int *v = new int;
    hipDeviceGetAttribute(v, hipDeviceAttributeMaxThreadsPerBlock, 0);
    cout << "hipDeviceAttributeMaxThreadsPerBlock = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    cout << "hipDeviceAttributeMaxSharedMemoryPerBlock = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeTotalConstantMemory, 0);
    cout << "hipDeviceAttributeTotalConstantMemory = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeWarpSize, 0);
    cout << "hipDeviceAttributeWarpSize = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMaxPitch, 0);
    cout << "hipDeviceAttributeMaxPitch = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMaxRegistersPerBlock, 0);
    cout << "hipDeviceAttributeMaxRegistersPerBlock = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeClockRate, 0);
    cout << "hipDeviceAttributeClockRate = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeAsyncEngineCount, 0);
    cout << "hipDeviceAttributeAsyncEngineCount = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMultiprocessorCount, 0);
    cout << "hipDeviceAttributeMultiprocessorCount = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeKernelExecTimeout, 0);
    cout << "hipDeviceAttributeKernelExecTimeout = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeIntegrated, 0);
    cout << "hipDeviceAttributeIntegrated = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMaxRegistersPerBlock, 0);
    cout << "hipDeviceAttributeMaxRegistersPerBlock = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeCanMapHostMemory, 0);
    cout << "hipDeviceAttributeCanMapHostMemory = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeComputeMode, 0);
    cout << "hipDeviceAttributeComputeMode = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeConcurrentKernels, 0);
    cout << "hipDeviceAttributeConcurrentKernels = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMemoryClockRate, 0);
    cout << "hipDeviceAttributeMemoryClockRate = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMemoryBusWidth, 0);
    cout << "hipDeviceAttributeMemoryBusWidth = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeL2CacheSize, 0);
    cout << "hipDeviceAttributeL2CacheSize = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    cout << "hipDeviceAttributeMaxThreadsPerMultiProcessor = " << *v << endl;

    hipDeviceGetAttribute(v, hipDeviceAttributeStreamPrioritiesSupported, 0);
    cout << "hipDeviceAttributeStreamPrioritiesSupported = " << *v << endl;

    hipStream_t stream;
    //hipStreamCreate(&stream);
    hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, *lp);
    hipStream_t hpStream;
    hipStreamCreateWithPriority(&hpStream, hipStreamNonBlocking, *hp);

    //cpu_data = new float*[nx];
    cpu_data = (float*) malloc(nx*ny*sizeof(float));
    for(int i = 0; i < nx*ny; i++)
    {
        cpu_data[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX); 
    }
    cout << "Init Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC/1000) 
         << " ms" << endl;

    // Copy to the GPU
    start = std::clock();
    float *gpu_data1;
    float *gpu_data2;
    hipMalloc(&gpu_data1, nx*ny*sizeof(float));
    hipMemcpyAsync(gpu_data1, cpu_data, nx*ny*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMalloc(&gpu_data2, nx*ny*sizeof(float));
    //hipDeviceSynchronize();
    cout << "Copy Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC/1000) 
         << " ms" << endl;

    dim3 dimGrid(1024);
    dim3 dimBlock(1024);

    // Start launching the kernel
    start = std::clock();
    for(int i = 0; i < simcycles; i++)
    {
        testKernel4<<<dimGrid, dimBlock, 0, stream>>>(gpu_data1, gpu_data2);
        //hipDeviceSynchronize();
        testKernel4r<<<dimGrid, dimBlock, 0, stream>>>(gpu_data1, gpu_data2);
        //hipDeviceSynchronize();
    }
    
    testKernelInject<<<1,1,0,hpStream>>>(gpu_data1);
    hipStreamSynchronize(hpStream);
    cout << "HP Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC/1000) 
         << " ms" << endl;

    hipDeviceSynchronize();
    cout << "Kernel Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC/1000) 
         << " ms" << endl;

    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
    {
        printf("Error: %s\n\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
