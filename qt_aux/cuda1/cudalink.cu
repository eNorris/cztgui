#include "hip/hip_runtime.h"

#include "cudalink.h"

float **init_gpu(int nx, int ny, double *cpu_data)
{
    if(nx <= 0 || ny <= 0)
        return NULL;

    std::cout << "Initializing GPU resources" << std::endl;

    // Find a gpu
    int devID = 0;
    hipDeviceProp_t props;
    //devID = findCudaDevice(0, 0);

    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    std::cout << "Device " << devID << ": " << props.name << " with compute "
         << props.major << "." << props.minor << " capability" << std::endl;

    std::clock_t start = std::clock();
    float *gpu_data1;
    float *gpu_data2;
    hipMalloc(&gpu_data1, nx*ny*sizeof(float));
    hipMemcpyAsync(gpu_data1, cpu_data, nx*ny*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&gpu_data2, nx*ny*sizeof(float));
    std::cout << "Copy Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC/1000)
         << " ms" << std::endl;

    float **gpu_datas = new float*[2];
    gpu_datas[0] = gpu_data1;
    gpu_datas[1] = gpu_data2;

    //double *gpu_data;
    //gpu_data = new double*[nx];
    //data_cpu = new double*[nx];
    //for(int i = 0; i < nx; i++)
    //{
    //    gpu_data[i] = new double[ny];
        //prevdata[i] = new double[ny];
    //    for(int j = 0;j < ny; j++)
    //    {
     //       gpu_data[i][j] = double(rand())/RAND_MAX;
            //prevdata[i][j] = data[i][j];
     //   }
    //}

    return gpu_datas;
}

int launch_testKernel(int &val)
{
    // Define the grid and blocks
    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 2);

    testKernel<<<dimGrid, dimBlock>>>(val);

    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
