#include "hip/hip_runtime.h"

#include <assert.h>

#include <hip/hip_runtime.h>

#include "cuda_common/inc/helper_functions.h"
#include "cuda_common/inc/hip/hip_runtime_api.h"
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#include <iostream>
#include <ctime>

#include "./cudakernel.h"

using namespace std;

int main()
{

    int devID;
    hipDeviceProp_t props;
    //devID = findCudaDevice(0, 0);

    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    cout << "Device " << devID << ": " << props.name << " with compute " 
         << props.major << "." << props.minor << " capability" << endl;

    //devID = 1;
    hipSetDevice(1);
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));
    cout << "Device " << devID << ": " << props.name << " with compute "
         << props.major << "." << props.minor << " capability" << endl;

    dim3 dimGrid(32, 32);
    dim3 dimBlock(16, 16, 4);

    //float q = 1000000.0;

    float *gpu_q;
    float *cpu_q = new float(100000.0);
    hipMalloc(&gpu_q, sizeof(float));
    hipMemcpy(gpu_q, cpu_q, sizeof(float), hipMemcpyHostToDevice); 

    std::clock_t start;
    start = std::clock();
    testKernel3<<<dimGrid, dimBlock>>>(gpu_q);
    hipDeviceSynchronize();
    cout << "Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC/1000) 
         << " ms" << endl;

    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
    {
        //hipError_t err = hipGetLastError();
        printf("Error: %s\n\n", hipGetErrorString(err));
    }
    //checkCudaErrors(hipGetDevice(&devID));

    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
